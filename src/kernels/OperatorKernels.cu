#include "hip/hip_runtime.h"
#include "OperatorKernels.h"
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void DeviceProceedVector(double* out, const double* in,
        OperatorElement* oper, const unsigned int dim, const unsigned int Nfibs)
{
    unsigned int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id < dim)
	{
        out[id] = 0;
        OperatorElement* m_oper = oper + 24 * id;
        for(unsigned int ind = 0; ind < 24; ind ++)
        {
            out[id] += m_oper[ind].coe * in[m_oper[ind].ind1] * in[m_oper[ind].ind2] * in[m_oper[ind].ind3];
        }
	}
}



void ProceedVector(double* out, const double* in,
        OperatorElement* oper, const unsigned int& dim, const unsigned int& Nfibs)
{
    DeviceProceedVector<<<dim/128 +1, 128>>>(out, in, oper, dim, Nfibs);
}

