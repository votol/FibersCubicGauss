#include "hip/hip_runtime.h"
#include "OperatorKernels.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include "ParameterDefines.h"

__global__ void DeviceMakeVector(hipDoubleComplex* vec, const hipDoubleComplex* cons,
		const hipDoubleComplex* ex, double * parameters,const unsigned int dim)
{
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id < dim)
	{
		vec[id].x = cons[id].x + ex[id].x * exp(-D*t); 
		vec[id].y = cons[id].y + ex[id].y * exp(-D*t);
	}
}



void MakeVector(hipDoubleComplex* vec, const hipDoubleComplex* cons,
		const hipDoubleComplex* ex, double * params,const unsigned int& dim)
{
	DeviceMakeVector<<<dim/128 +1, 128>>>(vec,cons,ex,params,dim);
}

