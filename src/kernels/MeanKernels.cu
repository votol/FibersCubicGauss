#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "MeanKernels.h"
#include "Indexes.h"


__global__ void DeviceMakeMean(double* out, const double* in, const unsigned int nfibs)
{
    unsigned int id = blockIdx.x*blockDim.x+threadIdx.x;
    if(id < nfibs)
    {
        out[id] = in[al_r(id)]*in[al_r(id)] + in[al_i(id)]*in[al_i(id)];
    }
}

void MakeMean(double* out, const double* in, const unsigned int& nfibs)
{
    DeviceMakeMean<<<nfibs/64 + 1, 64>>>(out, in, nfibs);
}
