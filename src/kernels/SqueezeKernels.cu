#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include "IntensityKernels.h"
#include "Indexes.h"


__global__ void DeviceMakeSqueeze(double* out, const double* in, const unsigned int nfibs, const unsigned int nalph)
{
    unsigned int id = blockIdx.x*blockDim.x+threadIdx.x;
    if(id < nfibs * nalph)
    {
        unsigned int n = id / nalph;
        unsigned int m = id % nalph;
        double alph = 2 * M_PI * double(m) / (double(nalph));
        out[id] = 0.5 + in[DI_r(n,n)] + in[Da_r(n,n)]*cos(alph) + in[Da_i(n,n)] * sin(alph);
    }
}

void MakeSqueeze(double* out, const double* in, const unsigned int& Nfibs, const unsigned int& nalph)
{
    DeviceMakeSqueeze<<<Nfibs * nalph/64 + 1, 64>>>(out, in, Nfibs, nalph);
}
