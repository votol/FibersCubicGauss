#include "hip/hip_runtime.h"
#include <math.h>
#include "EntanglementKernels.h"
#include "Indexes.h"

//one frequence
#define x1 (vec[DI_r(n,n)] + vec[Da_r(n,n)] + 0.5)
#define x2 (vec[DI_r(n,n)] - vec[Da_r(n,n)] + 0.5)
#define x3 (vec[DI_r(m,m)] + vec[Da_r(m,m)] + 0.5)
#define x4 (vec[DI_r(m,m)] - vec[Da_r(m,m)] + 0.5)
#define x5 (vec[Da_i(n,n)])
#define x6 (vec[DI_r(n,m)] + vec[Da_r(n,m)])
#define x7 ((n==m)?0:(vec[DI_i(n,m)]) + vec[Da_i(n,m)])
#define x8 ((n==m)?0:(-vec[DI_i(n,m)]) + vec[Da_i(n,m)])
#define x9 (vec[DI_r(n,m)] - vec[Da_r(n,m)])
#define x10 (vec[Da_i(m,m)])

__global__ void DeviceMakeEntanglementMatrix(const double* vec, double* matr, const unsigned int nfibs)
{
	int id = blockIdx.x*blockDim.x+threadIdx.x;
    int shift = nfibs*nfibs;
	if(id < shift)
	{
        int m = id/nfibs;
        int n = id%nfibs;
        if(n > m)
        {
            m += n;
            n = m - n;
            m = m - n;
        }
        double tmp = - x5 * x5 - x10 * x10 + x1*x2 + x3*x4 - 2*x6*x9 + 2*x7*x8;
        tmp = sqrt(2.0*(tmp - sqrt(tmp*tmp - 4.0 * (
                     x5*x5*x10*x10 - x3*x4*x5*x5 + 2*x4*x5*x6*x8 - 2*x5*x6*x9*x10 - 2*x5*x7*x8*x10 + 2*x3*x5*x7*x9 + x6*x6*x9*x9 - x2*x4*x6*x6 - 2*x6*x7*x8*x9 + 2*x2*x6*x7*x10 + x7*x7*x8*x8 - x2*x3*x7*x7 - x1*x4*x8*x8 + 2*x1*x8*x9*x10 - x1*x3*x9*x9 - x1*x2*x10*x10 + x1*x2*x3*x4))) );
        if(tmp > 0.0 && tmp< 1.0 && m!=n)
            matr[id] = -log(tmp)/log(2.0);
        else
            matr[id] = 0.0;
		
	}
}

void MakeEntanglementMatrix(const double* vec, double* matr, const unsigned int& dim)
{
    DeviceMakeEntanglementMatrix<<<dim*dim/64 +1,64>>>(vec,matr,dim);
}

