#include "hip/hip_runtime.h"
#include <math.h>
#include "EntanglementKernels.h"

//one frequence
#define x1 (vec[m*dim+m].x + 0.5)
#define x2 (vec[n*dim+n].x + 0.5)
#define x3 (vec[m*dim+n].x)
#define x4 (vec[m*dim+n].y)

//different frequence
#define y1 (vec[m*dim+m].x)
#define y2 (vec[shift + n*dim+n].x)
#define y3 (vec[2*shift + m*dim+n].x)
#define y4 (vec[2*shift + m*dim+n].y)


__global__ void DeviceMakeEntanglementMatrix1(const hipDoubleComplex* vec, double* matr, const unsigned int dim)
{
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	int shift = dim*dim;
	if(id < shift)
	{
		int m = id/dim;
		int n = id%dim;
		double tmp = x1*x1 + x2*x2 - 2*x3*x3 - 2*x4*x4;
		tmp = sqrt(2.0*(tmp - sqrt(tmp*tmp - 4.0 * ((x3*x3 + x4*x4 - x1*x2)*(x3*x3 + x4*x4 - x1*x2))) ));
		if(tmp > 0.0 && tmp< 1.0 && m!=n)
			matr[id] = -log(tmp)/log(2.0);
		else
			matr[id] = 0.0;
		
	}
}

__global__ void DeviceMakeEntanglementMatrix2(const hipDoubleComplex* vec, double* matr, const unsigned int dim)
{
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	int shift = dim*dim;
	if(id < shift)
	{
		int m = id/dim;
		int n = id%dim;
		double tmp = y1*y1 + y2*y2 + 2*y3*y3 + 2*y4*y4;
		tmp = sqrt(2.0*(tmp - sqrt(tmp*tmp - 4.0 * ((y3*y3 + y4*y4 - y1*y2)*(y3*y3 + y4*y4 - y1*y2))) ));
		if(tmp > 0.0 && tmp< 1.0)
			matr[id] = -log(tmp)/log(2.0);
		else
			matr[id] = 0.0;
		
	}
}

void MakeEntanglementMatrix1(const hipDoubleComplex* vec, double* matr, const unsigned int& dim)
{
	DeviceMakeEntanglementMatrix1<<<dim*dim/64 +1,64>>>(vec,matr,dim);
}

void MakeEntanglementMatrix2(const hipDoubleComplex* vec, double* matr, const unsigned int& dim)
{
	DeviceMakeEntanglementMatrix2<<<dim*dim/64 +1,64>>>(vec,matr,dim);
}
