#include "hip/hip_runtime.h"
#include <math.h>
#include "EntanglementKernels.h"

//one frequence
#define x1 (vec[m*dim+m].x + 0.5)
#define x2 (vec[n*dim+n].x + 0.5)
#define x3 (vec[m*dim+n].x)
#define x4 (vec[m*dim+n].y)

//different frequence
#define y1 (vec[m*dim+m].x)
#define y2 (vec[shift + n*dim+n].x)
#define y3 (vec[2*shift + m*dim+n].x)
#define y4 (vec[2*shift + m*dim+n].y)


__global__ void DeviceMakeEntanglementMatrix(const double* vec, double* matr, const unsigned int dim)
{
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	int shift = dim*dim;
	if(id < shift)
	{
		int m = id/dim;
		int n = id%dim;
        //double tmp = x1*x1 + x2*x2 - 2*x3*x3 - 2*x4*x4;
        //tmp = sqrt(2.0*(tmp - sqrt(tmp*tmp - 4.0 * ((x3*x3 + x4*x4 - x1*x2)*(x3*x3 + x4*x4 - x1*x2))) ));
        //if(tmp > 0.0 && tmp< 1.0 && m!=n)
        //	matr[id] = -log(tmp)/log(2.0);
        //else
            matr[id] = 0.0;
		
	}
}

void MakeEntanglementMatrix(const double* vec, double* matr, const unsigned int& dim)
{
    DeviceMakeEntanglementMatrix<<<dim*dim/64 +1,64>>>(vec,matr,dim);
}

