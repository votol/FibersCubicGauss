#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "IntensityKernels.h"
#include "Indexes.h"


__global__ void DeviceMakeIntensity(double* out, const double* in, const unsigned int nfibs)
{
    unsigned int id = blockIdx.x*blockDim.x+threadIdx.x;
    if(id < 256)
    {
        out[id] = 6.0;//in[DI_r(id,id)] + in[al_r(id)]*in[al_r(id)] + in[al_i(id)]*in[al_i(id)];
    }
}

void MakeIntensity(double* out, const double* in, const unsigned int& nfibs)
{
    DeviceMakeIntensity<<<nfibs/64 + 1, 64>>>(out, in, nfibs);
}
